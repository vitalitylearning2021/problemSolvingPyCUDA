#include "hip/hip_runtime.h"
// nvcc -Xcompiler -fPIC -shared -o testSortByKeyDLL.dll testSortByKeyDLL.cu

#include <hip/hip_runtime.h>

#include <thrust/sort.h>
#include <thrust/device_ptr.h>

/******************/
/* ERROR CHECKING */
/******************/
#define cudaCHECK(ans) { checkAssert((ans), __FILE__, __LINE__); }
inline void checkAssert(hipError_t errorCode, const char *file, int line, bool abort = true) {
	if (errorCode != hipSuccess) {
		fprintf(stderr, "Check assert: %s %s %d\n", hipGetErrorString(errorCode), file, line);
		if (abort) exit(errorCode);
	}
}

#define DELLEXPORT extern "C" __declspec(dllexport)

/**************************/
/* CUDASORTBYKEY FUNCTION */
/**************************/
DELLEXPORT void cudaSortByKey(int *h_key_in, float2 *h_val_in, int *h_key_out, float2 *h_val_out, const int N){
	
	int *d_key_in; 		cudaCHECK(hipMalloc(&d_key_in,   N * sizeof(int)));
	int *d_key_out; 	cudaCHECK(hipMalloc(&d_key_out,  N * sizeof(int)));
	float2 *d_val_in; 	cudaCHECK(hipMalloc(&d_val_in,   N * sizeof(float2)));
	float2 *d_val_out; 	cudaCHECK(hipMalloc(&d_val_out,  N * sizeof(float2)));

	cudaCHECK(hipMemcpy(d_key_in, h_key_in, N * sizeof(int),    hipMemcpyHostToDevice));
	cudaCHECK(hipMemcpy(d_val_in, h_val_in, N * sizeof(float2), hipMemcpyHostToDevice));
	
	thrust::device_ptr<int> 	d_key_in_dev_ptr = thrust::device_pointer_cast(d_key_in);	
	thrust::device_ptr<float2> 	d_val_in_dev_ptr = thrust::device_pointer_cast(d_val_in);	

	thrust::sort_by_key(d_key_in_dev_ptr, d_key_in_dev_ptr + N, d_val_in_dev_ptr);

	cudaCHECK(hipMemcpy(h_key_out, d_key_in, N * sizeof(int),    hipMemcpyDeviceToHost));
	cudaCHECK(hipMemcpy(h_val_out, d_val_in, N * sizeof(float2), hipMemcpyDeviceToHost));

	cudaCHECK(hipFree(d_key_in));
	cudaCHECK(hipFree(d_val_in));
	cudaCHECK(hipFree(d_key_out));
	cudaCHECK(hipFree(d_val_out));
}