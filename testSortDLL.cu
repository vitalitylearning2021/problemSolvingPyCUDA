// nvcc -Xcompiler -fPIC -shared -o testSortDLL.dll testSortDLL.cu

#include <hip/hip_runtime.h>

#include <thrust/sort.h>
#include <thrust/device_ptr.h>

/******************/
/* ERROR CHECKING */
/******************/
#define cudaCHECK(ans) { checkAssert((ans), __FILE__, __LINE__); }
inline void checkAssert(hipError_t errorCode, const char *file, int line, bool abort = true) {
	if (errorCode != hipSuccess) {
		fprintf(stderr, "Check assert: %s %s %d\n", hipGetErrorString(errorCode), file, line);
		if (abort) exit(errorCode);
	}
}

#define DELLEXPORT extern "C" __declspec(dllexport)

/*********************/
/* CUDASORT FUNCTION */
/*********************/
DELLEXPORT void cudaSort(int *h_in, int *h_out, const int N){
	
	int *d_in; cudaCHECK(hipMalloc(&d_in,  N * sizeof(int)));

	cudaCHECK(hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice));
	
	thrust::device_ptr<int> d_in_dev_ptr = thrust::device_pointer_cast(d_in);	

	thrust::sort(d_in_dev_ptr, d_in_dev_ptr + N);

	cudaCHECK(hipMemcpy(h_out, d_in, N * sizeof(int), hipMemcpyDeviceToHost));

	cudaCHECK(hipFree(d_in));
}
